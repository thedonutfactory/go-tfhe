#include <stdio.h>
#include <hip/hip_runtime.h>
 

__global__ void vecmul(int32_t *A, int32_t* B, int32_t *C, int size)
{
    // Row and Column indexes: 
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    // Are they bellow the maximum?
    if (col < size && row < size) {
       int32_t result = 0;
       for(int ix=0;ix<size;ix++) {
          result += A[row*size+ix]*B[ix*size+col];
       }
       C[row*size+col] = result;
    }
}

extern "C" {

    void maxmul(int32_t *A, int32_t* B, int32_t *C, int size) {

        int total = size*size;

        // Allocate device memory:
        int32_t* gpu_A;
        int32_t* gpu_B;
        int32_t* gpu_C;
        int msize = total * sizeof(int32_t);
        hipMalloc((void**)&gpu_A, msize);
        hipMemcpy(gpu_A,A,msize,hipMemcpyHostToDevice);
        hipMalloc((void**)&gpu_B, msize);
        hipMemcpy(gpu_B,B,msize,hipMemcpyHostToDevice);
        hipMalloc((void**)&gpu_C,msize);

        // Blocks & grids:
        dim3 blocks(size,size);
        dim3 grid(1,1);

        // Call the kernel:
        vecmul<<<grid,blocks>>>(gpu_A,gpu_B,gpu_C,size);

        // Get the result Matrix:
        hipMemcpy(C,gpu_C,msize,hipMemcpyDeviceToHost);

        //Free device matrices
        hipFree(gpu_A);
        hipFree(gpu_B);
        hipFree(gpu_C);
    }

}