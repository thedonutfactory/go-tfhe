
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <vector>
#include <cmath>

#include <stdio.h>

const auto BATCH = 1;

__global__ void ComplexPointwiseMulAndScale(hipfftComplex *a, hipfftComplex *b, int size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    float scale = 1.0f / (float)size;
    hipfftComplex c;
    for (int i = threadID; i < size; i += numThreads)
    {
        c = hipCmulf(a[i], b[i]);
        b[i] = make_hipFloatComplex(scale*hipCrealf(c), scale*hipCimagf(c));
    }
}

__global__ void ConvertToInt(hipfftReal *a, int size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    auto b = (int*)a;
    for (int i = threadID; i < size; i += numThreads)
        b[i] = static_cast<int>(round(a[i]));
}

std::vector<int> multiply(const std::vector<float> &a, const std::vector<float> &b)
{
    const auto NX = a.size();
    hipfftHandle plan_a, plan_b, plan_c;
    hipfftComplex *data_a, *data_b;
    std::vector<int> c(a.size() + 1);
    c[0] = 0;

    //Allocate graphics card memory and initialize, assuming sizeof(int)==sizeof(float), sizeof(cufftComplex)==2*sizeof(float)
    hipMalloc((void**)&data_a, sizeof(hipfftComplex) * (NX / 2 + 1) * BATCH);
    hipMalloc((void**)&data_b, sizeof(hipfftComplex) * (NX / 2 + 1) * BATCH);
    hipMemcpy(data_a, a.data(), sizeof(float) * a.size(), hipMemcpyHostToDevice);
    hipMemcpy(data_b, b.data(), sizeof(float) * b.size(), hipMemcpyHostToDevice);
    if (hipGetLastError() != hipSuccess) { fprintf(stderr, "Cuda error: Failed to allocate\n"); return c; }

    if (hipfftPlan1d(&plan_a, NX, HIPFFT_R2C, BATCH) != HIPFFT_SUCCESS) { fprintf(stderr, "CUFFT error: Plan creation failed"); return c; }
    if (hipfftPlan1d(&plan_b, NX, HIPFFT_R2C, BATCH) != HIPFFT_SUCCESS) { fprintf(stderr, "CUFFT error: Plan creation failed"); return c; }
    if (hipfftPlan1d(&plan_c, NX, HIPFFT_C2R, BATCH) != HIPFFT_SUCCESS) { fprintf(stderr, "CUFFT error: Plan creation failed"); return c; }

    //Converting A(x) to Frequency Domain
    if (hipfftExecR2C(plan_a, (hipfftReal*)data_a, data_a) != HIPFFT_SUCCESS)
    {
        fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
        return c;
    }

    //Converting B(x) to Frequency Domain
    if (hipfftExecR2C(plan_b, (hipfftReal*)data_b, data_b) != HIPFFT_SUCCESS)
    {
        fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
        return c;
    }

    //Point multiplication
    ComplexPointwiseMulAndScale<<<NX / 256 + 1, 256>>>(data_a, data_b, NX);

    //Converting C(x) back to time domain
    if (hipfftExecC2R(plan_c, data_b, (hipfftReal*)data_b) != HIPFFT_SUCCESS)
    {
        fprintf(stderr, "CUFFT error: ExecC2R Forward failed");
        return c;
    }

    //Converting the results of floating-point numbers to integers
    ConvertToInt<<<NX / 256 + 1, 256>>>((hipfftReal*)data_b, NX);

    if (hipDeviceSynchronize() != hipSuccess) 
    {
        fprintf(stderr, "Cuda error: Failed to synchronize\n");
        return c;
    }

    hipMemcpy(&c[1], data_b, sizeof(float) * b.size(), hipMemcpyDeviceToHost);

    hipfftDestroy(plan_a);
    hipfftDestroy(plan_b);
    hipfftDestroy(plan_c);
    hipFree(data_a);
    hipFree(data_b);
    return c;
}


int main(int argc, char **argv) 
{
    //Set base
    const auto base = 10;

    //999 * 9
    std::vector<float> a{ 0, 9, 9, 9 }; 
    std::vector<float> b{ 0, 0, 0, 9 };

    auto c = multiply(a, b);

    for (auto i : c)
        printf("%d ", i);
    printf("\n");

    //Processing carry
    for (int i = c.size() - 1; i > 0; i--)
    {
        if (c[i] >= base)
        {
            c[i - 1] += c[i] / base;
            c[i] %= base;
        }
    }

    //Remove excess zeros
    c.pop_back();
    auto i = 0;
    if (c[0] == 0)
        i++;

    //To output the final result, we need to change the mode of output, such as the decimal system is "% 2d" and the decimal system is "% 3d".
    for (; i < c.size(); i++)
        printf("%d", c[i]);
    printf("\n");

    return 0;
}