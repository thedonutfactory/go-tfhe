
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>

//void multiplyWithCuda(long *c, const long *a, const long *b, unsigned int size);
//void multiplyWithCudaKaratsuba(long *c, const long *a, const long *b, unsigned int size);

__device__ void multiplyKernelKaratsubaRec(long *z, const long *x, const long *y, unsigned int size)
{
	const long *a, *b, *c, *d;
	long *ab, *ac;
	long *bd, *cd;
	long *adbc;


	if (size <= 1)
	{
		z[0] = x[0] * y[0];
	}
	else
	{
		int half = (int)size / 2;

		ab = (long*)malloc(half * sizeof(long));
		ac = (long*)malloc(half * sizeof(long));
		cd = (long*)malloc(half * sizeof(long));
		bd = (long*)malloc(half * sizeof(long));
		adbc = (long*)malloc(half * sizeof(long));

		a = x;
		b = x + half;

		c = y;
		d = y + half;

		multiplyKernelKaratsubaRec(ac, a, c, half);
		multiplyKernelKaratsubaRec(bd, b, d, size - half);

		int i = 0;
		for (i = 0; i < half; i++)
		{
			ab[i] = a[i] + b[i];
			cd[i] = c[i] + d[i];
		}

		multiplyKernelKaratsubaRec(adbc, ab, cd, half);

		for (i = 0; i < half; i++)
		{
			z[i] = adbc[i] - ac[i] - bd[i];
		}
	}
}

__global__ void multiplyKernelKaratsuba(long *z, const long *x, const long *y, unsigned int size)
{
	multiplyKernelKaratsubaRec(z, x, y, size);
}

__global__ void multiplyKernel(long *c, const long *a, const long *b, unsigned int size)
{
    int i = threadIdx.x;
	c[i] = 0;
	for (auto x = 0; x < size; x++)
    {
	    for (auto y = 0; y < size; y++)
	    {
		    if (x + y == i)
		    {
				c[i] += a[x] * b[y];
		    }
	    }
    }
}

extern "C" {
	void multiplyWithCudaKaratsuba(long *c, const long *a, const long *b, unsigned int size)
	{
		long *dev_a = nullptr;
		long *dev_b = nullptr;
		long *dev_c = nullptr;

		hipSetDevice(0);

		hipMalloc(&dev_c, 2 * size * sizeof(long));
		hipMalloc(&dev_a, size * sizeof(long));
		hipMalloc(&dev_b, size * sizeof(long));

		hipMemcpy(dev_a, a, size * sizeof(long), hipMemcpyHostToDevice);
		hipMemcpy(dev_b, b, size * sizeof(long), hipMemcpyHostToDevice);

		int thread_num = 2 * size;
		multiplyKernelKaratsuba <<<1, thread_num >>> (dev_c, dev_a, dev_b, size);

		hipDeviceSynchronize();

		hipMemcpy(c, dev_c, 2 * size * sizeof(long), hipMemcpyDeviceToHost);

		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);
	}

	void multiplyWithCuda(long *c, const long *a, const long *b, unsigned int size)
	{
		long *dev_a = nullptr;
		long *dev_b = nullptr;
		long *dev_c = nullptr;

		hipSetDevice(0);

		hipMalloc(&dev_c, 2 * size * sizeof(long));
		hipMalloc(&dev_a, size * sizeof(long));
		hipMalloc(&dev_b, size * sizeof(long));
		
		hipMemcpy(dev_a, a, size * sizeof(long), hipMemcpyHostToDevice);
		hipMemcpy(dev_b, b, size * sizeof(long), hipMemcpyHostToDevice);

		int thread_num = 2 * size;
		multiplyKernel<<<1, thread_num>>>(dev_c, dev_a, dev_b, size);
		
		hipDeviceSynchronize();

		hipMemcpy(c, dev_c, 2 * size * sizeof(long), hipMemcpyDeviceToHost);

		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);

		printf("first val: %ld\n", c[0]);
	}

}

int main()
{
	const auto arraySize = 9;
	//long a[arraySize];
	//long b[arraySize];
	long c[2 * arraySize];

	long a[] = { 0, 9, 9, 9, 0, 9, 9, 9 };
	long b[] = { 0, 0, 0, 0, 0, 9, 9, 9 };

	for (auto i = 0; i < arraySize; i++) {
		//a[i] = rand() % 100;
		//b[i] = rand() % 100;
		c[i] = c[arraySize + i] = 0;
	}

    // Multiply polynomials in parallel.
	multiplyWithCuda(c, a, b, arraySize);
	
	for (auto i = 0; i < arraySize; i++) {
		printf("%ld, ", c[i]);
	}
	printf("\n");
}

int main2()
{
	srand(time(nullptr));
	
	const auto arraySize = 1024;
	long a[arraySize];
	long b[arraySize];
	long c[2 * arraySize];

	for (auto i = 0; i < arraySize; i++)
	{
		a[i] = rand() % 100;
		b[i] = rand() % 100;
		c[i] = c[arraySize + i] = 0;
	}

    // Multiply polynomials in parallel.
	time_t timeStart;
	time_t timeEnd;
	time(&timeStart);
	for (auto i = 0; i < 10000; i++)
		multiplyWithCuda(c, a, b, arraySize);
	time(&timeEnd);

	printf("time taken (normal) : %ld (%ld : %ld) \n", timeEnd - timeStart, timeStart, timeEnd);

	time(&timeStart);
	for (auto i = 0; i < 10000; i++)
		multiplyWithCudaKaratsuba(c, a, b, arraySize);
	time(&timeEnd);

	printf("time taken (karatsuba) : %ld (%ld : %ld) \n", timeEnd - timeStart, timeStart, timeEnd);


//	for (auto i = 0; i < arraySize; i++)
//	{
//		printf("%d ", a[i]);
//	}
//	printf("\n");
//
//	for (auto i = 0; i < arraySize; i++)
//	{
//		printf("%d ", b[i]);
//	}
//	printf("\n");
//
//    for (auto i = 0; i < 2 * arraySize; i++)
//    {
//		printf("%d ", c[i]);
//    }
//	printf("\n");

    hipDeviceReset();

    return 0;
}
